#include "hip/hip_runtime.h"
#include "clusterdata.h"

int SHAREDATA_ROWS = 16;

double* loadDatasetNumeric(Dataset* dataset);
void setFirstClusters(double* centroids, double* records, int k, int rows, int cols);


__device__ double euclideanDistance(double *record1, double *record2, int cols){
	
	double dist = 0.0f;
	int i;
	for(i = 1; i < cols; i++){
		dist += (record1[i]-record2[i]) * (record1[i]-record2[i]);
	}
	return sqrt(dist);
}

__global__ void findClosestClusters(double* centroids, int k, double* records, int rows, int cols){

	extern __shared__ double s_records[];

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int sharedIdx = threadIdx.y*cols + threadIdx.x;
	int recordIdx = idy*cols + threadIdx.x;
	if(idx >= cols || idy >= rows)
		return;
	s_records[sharedIdx] = records[recordIdx];
	__syncthreads();
	
	if(threadIdx.x==0){
//		printf("idy %d\n", idy);
		int closestCluster = (int)s_records[sharedIdx];
		double closestDistance = euclideanDistance(&s_records[sharedIdx], &centroids[closestCluster * cols], cols);
		double thisDistance;
		int i;
		for(i = 0; i < k; i++){
			thisDistance = euclideanDistance(&s_records[sharedIdx], &centroids[i * cols], cols);
			double diff = thisDistance - closestDistance;
//			printf("k %d, Record %d closest %d dist%lf check %d newDist %lf, dist%lf\n",k , idy, closestCluster, closestDistance, i, thisDistance, thisDistance-closestDistance);
			if(diff < 0){
//				printf("Record %d oldk %d newk %d olddist %lf newdist %lf\n", idy, closestCluster, i, closestDistance, thisDistance);
				closestDistance = thisDistance;
				s_records[sharedIdx] = i;
			}
		
		}
		records[idy*cols] = s_records[sharedIdx];	
	}
}


__device__ void calculateSSE(double* centroids, int k, double* records, int rows, int cols){

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idy = blockIdx.y * blockDim.y + threadIdx.y;
	int sharedIdx = threadIdx.y*cols + threadIdx.x;
	int recordIdx = idy*cols + threadIdx.x;
	if(idx >= cols || idy >= rows)
		return;
	s_records[sharedIdx] = records[recordIdx];
	__syncthreads();
}
/*
/ beginGpuCluster()	Sets up kernel and performs clustering
/			Records find cluster on GPU, SSE performed on GPU	
*/
double* beginGpuClustering(double* centroids, double *records, int k, int num_rows, int num_cols){

	hipSetDevice(3);
	hipDeviceReset();
	dim3 blockdim;
	dim3 griddim;
	size_t sharedsize;
	size_t clustersize;
	size_t recordsize; 

	blockdim.x = num_cols;
	blockdim.y = SHAREDATA_ROWS;
	griddim.x = 1;
	griddim.y =  ceil((float)num_rows/SHAREDATA_ROWS);
	sharedsize = SHAREDATA_ROWS * num_cols * sizeof(double);	
	clustersize = k*num_cols*sizeof(double);
	recordsize = num_rows*num_cols*sizeof(double);	

	double *d_centroids;
	double *d_records;
	
	int i,j;
	printf("\n\n");
	printf("Threads x %d Threads y %d Blocks x %d blocksy %d\n", blockdim.x, blockdim.y, griddim.x, griddim.y);
	hipMalloc((void**)&d_centroids, clustersize);
	hipMalloc((void**)&d_records, recordsize);
	hipMemcpy(d_centroids, centroids, clustersize, hipMemcpyHostToDevice);
	hipMemcpy(d_records, records, recordsize, hipMemcpyHostToDevice);
	
	findClosestClusters<<<griddim,blockdim, sharedsize>>>(d_centroids, k, d_records, num_rows, num_cols);
	
	hipMemcpy(centroids, d_centroids, clustersize, hipMemcpyDeviceToHost);
	hipMemcpy(records, d_records, recordsize, hipMemcpyDeviceToHost);

	for(i = 0; i < 20; i++){
		for(j = 0; j < num_cols; j++){
			printf("%f ", records[i*num_cols+j]);
		}
		printf("\n");
	}
	return NULL;
}

/*
/ clusterData()		Converts the dataset to numeric values and centroids
/					Clusters the records in the file into k centroids (gpu)
/					Uses squared error check for convergence (gpu)
*/
double* clusterData(Dataset* dataset, int k){

	double* centroids = (double*)calloc(k * (dataset->num_cols+1), sizeof(double));
	double* numeric_records = loadDatasetNumeric(dataset);
	setFirstClusters(centroids, numeric_records, k, dataset->num_rows-1, dataset->num_cols+1);	
	beginGpuClustering(centroids, numeric_records, k, dataset->num_rows-1, dataset->num_cols+1);	
	return centroids;
}

/**
/ loadDatasetNumeric()	Converts the set of strings to double values
/						Loads a flattened double* array with record values
*/
double* loadDatasetNumeric(Dataset* dataset){

	int thisrow, thiscol;
	int num_rows = dataset->num_rows;
	int num_cols = dataset->num_cols;
	char** records = dataset->records;
	double *DATASET_NUMERIC;
	
	DATASET_NUMERIC = (double*)calloc(num_rows * (num_cols+1), sizeof(double));

	for(thisrow = 1; thisrow < num_rows; thisrow++){

		for(thiscol = 0; thiscol < num_cols+1; thiscol++){
			int numericIdx = (thisrow-1)*(num_cols+1)+thiscol;
			int recordIdx = thisrow*num_cols+thiscol-1;
			if(thiscol == 0)
				DATASET_NUMERIC[numericIdx] = 0;
			else{
				sscanf(records[recordIdx], "%lf", (double*)&DATASET_NUMERIC[numericIdx]);
			}
		}
	}
	return DATASET_NUMERIC;
}

/**
/ setFirstClusters() 	Sets the initial cluster numbers and records
						Assigned a random record value to each cluster
*/
void setFirstClusters(double* centroids, double* records, int k, int rows, int cols){

	int i,j, r;
	time_t t;
	int *rands = (int*)calloc(k, sizeof(int));
	srand((unsigned) time(&t));
	int randn;

	// set initial cluster numbers
	for(i = 0; i < k; i++){
		randn = abs(rand() % (rows-1));
		int done = false;
		
		while(!done){
			done = true; 
			for(r = 0; r < k; r++){
				if(randn == rands[r]){
					done = false;
					randn = abs(rand() % (rows-1));
				}
			}
		}
		
		printf("rows %d cols %d i %d randn %d\n", rows, cols, i, randn);	
		rands[i] = randn;
		
		for(j = 0; j < cols; j++){
			if(j==0){
				centroids[i*cols] = i;
				records[randn * cols] = i;
				printf("records[%d]: %f  centroid: %d\n", randn, records[randn*cols], i);
			}
			else
				centroids[i*cols+j] = records[randn * cols + j];
		}
	}
	free(rands);
}
